#include "hip/hip_runtime.h"
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <time.h>

#include "radray.h"

#define MAX_THREADS 1024


/* Function used to check if the point is inside the cube */
__device__ int point_in_polygon_dev(point2d *limits, int N, float minz, float maxz ,point3d p){
    int inside=0;
    point2d p1,p2;
    p1 = limits[0];

    for(int i=1;i<N;i++){
        p2=limits[i%N];
        printf("%f %f %f\n", p.z, minz, maxz);
        if((p.y>min(p1.y,p2.y)) &&(p.y<=max(p1.y,p2.y))&&(p.x<=max(p1.x,p2.x))&&(p1.y != p2.y)){
            if (p.x<(p.y-p1.y)*(p2.x-p1.x)/(p2.y-p1.y)+p1.x && p.z>=minz && p.z<=maxz){
                inside=!inside;
            }
        }
        p1=p2;
    }
    return inside;
}


/* Kernel lauched to initialize the points of the cube hit by the ray. Each threads has a specific point in the area of the cube, if the point is contained in the cube itself (not only 
in the area surrounding the cube) the point is initialized.
TODO: maybe create struct for the parameters or something like that to reduce the lenght of function call
*/
__global__ void initialize_points(point2d *limits, energy_point *points, int x_amt, int y_amt, int z_amt, float minx, float miny, float minz, int dx, int dy, int dz, int N, float maxz) {

    int tid = blockDim.x*blockIdx.x+threadIdx.x;
    int x = tid%x_amt;
    int y = (tid/x_amt)%y_amt;
    int z = tid/(x_amt*y_amt);
    point3d t;

    // with this condition we exclude the thread in excess
    if (z<z_amt) {
        t.x = minx + x * dx;
        t.y = miny + y * dy;
        t.z = minz + z * dz;
        //printf("%f %f %f\n", t.x, t.y, t.z);
        if (point_in_polygon_dev(limits, N, minz, maxz, t)) {
            //printf("%d %d %d %f %f\n", z, z_amt, dz, t.z, minz);
            //printf("%f %f %f\n", t.x, t.y, t.z);
            points[tid].pos = t;
            points[tid].energy[0] = 0;
            points[tid].energy[N_STEPS] = 0;
        }
    }

}


int read_input(char* inpath,cube cubes[],point3d* CUBE_GLOBAL_MAX, point3d* CUBE_GLOBAL_MIN){
    FILE* fin;
    cube t;
    int cube_number=0,i;

    //Open the file
    fin = fopen(inpath, "r");
    if(fin==NULL){
        printf("Unable to read the file!");
        exit(1);
    }
    /***
     * Read file, format for each boundary:                 \n
     * N    layer                                           \n
     * minz maxz                                            \n
     * x    y   (repeated N times)                          \n
     * Calculates minx and miny of the cube Bounding Box    \n
     * Calculates Global bounding box                       \n
     */
    while(fscanf(fin, "%d %d", &t.N, &t.layer_n) != EOF) {
        t.limits = (point2d *) malloc(t.N * sizeof(point2d));
        t.limits = (point2d *) malloc(t.N * sizeof(point2d));
        fscanf(fin, "%f %f",&t.min.z,&t.max.z);
        for (i = 0; i < t.N; i++) {
            fscanf(fin, "%f %f", &(t.limits[i].x), &t.limits[i].y);
            if (i == 0) {
                t.min.x = t.limits[0].x;
                t.min.y = t.limits[0].y;
                t.max.x = t.limits[0].x;
                t.max.y = t.limits[0].y;
            } else {
                t.max.x = t.limits[i].x > t.max.x ? t.limits[i].x : t.max.x;
                t.min.x = t.limits[i].x < t.min.x ? t.limits[i].x : t.min.x;
                t.max.y = t.limits[i].y > t.max.y ? t.limits[i].y : t.max.y;
                t.min.y = t.limits[i].y < t.min.y ? t.limits[i].y : t.min.y;
            }
        }
        if (t.max.x > CUBE_GLOBAL_MAX->x) { CUBE_GLOBAL_MAX->x = t.max.x; }  // computes global max and min, in which the ray must pass
        if (t.max.y > CUBE_GLOBAL_MAX->y) { CUBE_GLOBAL_MAX->y = t.max.y; }
        if (t.max.z > CUBE_GLOBAL_MAX->z) { CUBE_GLOBAL_MAX->z = t.max.z; }
        if (t.min.x < CUBE_GLOBAL_MIN->x) { CUBE_GLOBAL_MIN->x = t.min.x; }
        if (t.min.y < CUBE_GLOBAL_MIN->y) { CUBE_GLOBAL_MIN->y = t.min.y; }
        if (t.min.z < CUBE_GLOBAL_MIN->z) { CUBE_GLOBAL_MIN->z = t.min.z; }
        t.points=NULL;
        cubes[cube_number] = t;
        cube_number++;
    }
    fclose(fin);
    return cube_number;
}

int cube_contains_point(cube cu, point3d p){
    if(p.x>=cu.min.x && p.y>=cu.min.y && p.z>=cu.min.z
    && p.x<=cu.max.x && p.y<=cu.max.y && p.z<=cu.max.z){
        return 1;
    }
    return 0;
}

int point_in_polygon(cube poly,point3d p){
    int inside=0;
    point2d p1,p2;
    p1=poly.limits[0];
    for(int i=1;i<poly.N;i++){
        p2=poly.limits[i%poly.N];
        printf("%f %f %f\n", p.z, poly.min.z, poly.max.z);
        if((p.y>min(p1.y,p2.y)) &&(p.y<=max(p1.y,p2.y))&&(p.x<=max(p1.x,p2.x))&&(p1.y != p2.y)){
            if (p.x<(p.y-p1.y)*(p2.x-p1.x)/(p2.y-p1.y)+p1.x && p.z>=poly.min.z && p.z<=poly.max.z){
                inside=!inside;
            }
        }
        p1=p2;
    }
    return inside;
}

int cube_contains_ray(cube cu, ray r) {
    point3d ray_pos = r.start;
    for (ray_pos.z = cu.max.z; ray_pos.z > cu.min.z; ray_pos.z+=r.delta.z) {    //rendere for da cu.min.z a cu.max.z
        if (point_in_polygon(cu, ray_pos)) return 1;
        ray_pos.x += r.delta.x;
        ray_pos.y += r.delta.y;
        //ray_pos.z += r.delta.z;
    }
    return 0;
}

ray rand_ray(point3d bound_min, point3d bound_max, energy_type profile) {     //randomly generates the trajectory of a ray given the box bounds
    ray r;
    r.profile = profile;
    r.start.x = rand_unit() * (bound_max.x - bound_min.x) + bound_min.x;
    r.start.y = rand_unit() * (bound_max.y - bound_min.y) + bound_min.y;
    r.start.z = bound_max.z;
    r.end.x = rand_unit() * (bound_max.x - bound_min.x) + bound_min.x;
    r.end.y = rand_unit() * (bound_max.y - bound_min.y) + bound_min.y;
    r.end.z = bound_min.z;
    r.delta.x = (r.end.x - r.start.x) / N_STEPS;
    r.delta.y = (r.end.y - r.start.y) / N_STEPS;
    r.delta.z = (r.end.z - r.start.z) / N_STEPS;
    r.steps = N_STEPS;
    generate_energy_profile(&r);
    return r;
}

ray fixed_ray(point3d start, point3d end, energy_type profile) {     //generates the trajectory of a ray given the start and end positions
    ray r;
    r.profile = profile;
    r.start = start;
    r.end = end;
    r.delta.x = (r.end.x - r.start.x) / N_STEPS;
    r.delta.y = (r.end.y - r.start.y) / N_STEPS;
    r.delta.z = (r.end.z - r.start.z) / N_STEPS;
    r.steps = N_STEPS;
    generate_energy_profile(&r);
    return r;
}

void generate_energy_profile(ray *ray) {    //Aggiungere scalamento in base alla lunghezza del raggio
    switch (ray->profile) {
        case Bragg:
            //TODO
            for (int i = 0; i < ENERGY_CURVE_SIZE; i++) {
                ;
            }
            break;
        
        case Linear:
            for (int i = 0; i < ENERGY_CURVE_SIZE; i++) {
                ray->energy_curve[i] = 1 - i/ENERGY_CURVE_SIZE * i;
            }
            break;
        
        case Constant:
        default:
            for (int i = 0; i < ENERGY_CURVE_SIZE; i++) {
                ray->energy_curve[i] = 1;
            }
            break;
    }
    return;
}

void generate_points_by_amount(cube *curr_cube, int amount){  //generates amount points on a grid in each box
    /***
     * Can be improved by generating only points in the boundary or only points near the ray.
     * */
    int cubroot = cbrtf(MAX_POINTS);
    point3d t;
    int dx = (curr_cube->max.x - curr_cube->min.x) / cubroot;
    int dy = (curr_cube->max.y - curr_cube->min.y) / cubroot;
    int dz = (curr_cube->max.z - curr_cube->min.z) / cubroot;
    int cnt=0;
    curr_cube->points = (energy_point *) malloc(MAX_POINTS * sizeof(energy_point));
    for(int i = 0; i < cubroot; i++){
        for(int j = 0; j < cubroot; j++){
            for(int k = 0; k < cubroot; k++){
                t.x = curr_cube->min.x + i * dx;
                t.y = curr_cube->min.y + j * dy;
                t.z = curr_cube->min.z + k * dz;
                curr_cube->points[cnt].pos = t;
                curr_cube->points[cnt].energy[0] = 0;
                curr_cube->points[cnt].energy[N_STEPS] = 0;
                cnt++;
            }
        }
    }
    curr_cube->point_amt = cnt;
    return;
}

void generate_points_by_resolution(cube *curr_cube, point3d resolution){  //generates MAX_POINTS points on a grid in each box
    point3d t;
    int cnt = 0;
    int dx = resolution.x;
    int dy = resolution.y;
    int dz = resolution.z;
    int x_amt = (curr_cube->max.x - curr_cube->min.x) / dx;
    int y_amt = (curr_cube->max.y - curr_cube->min.y) / dy;
    int z_amt = (curr_cube->max.z - curr_cube->min.z) / dz;
    curr_cube->points = (energy_point *) malloc(x_amt * y_amt * z_amt * sizeof(energy_point));  //nvcc vuole il cast
    for(int i = 0; i < x_amt; i++){
        for(int j = 0; j < y_amt; j++){
            for(int k = 0; k < z_amt; k++){
                t.x = curr_cube->min.x + i * dx;
                t.y = curr_cube->min.y + j * dy;
                t.z = curr_cube->min.z + k * dz;
                //printf("%f %f %f\n", t.x, t.y, t.z);
                if (point_in_polygon(*curr_cube, t)) {
                    //printf("%d %d %d %f %f\n", k, z_amt, dz, t.z, curr_cube->min.z);
                    //printf("%f %f %f\n", t.x, t.y, t.z);
                    curr_cube->points[cnt].pos = t;
                    curr_cube->points[cnt].energy[0] = 0;
                    curr_cube->points[cnt].energy[N_STEPS] = 0;
                    cnt++;
                }
            }
        }
    }
    curr_cube->point_amt = cnt;
    return;
}


/* Generates the points inside a cube with a given resolution, it directly returns the pointer to the points vector in the GPU so that we don't need to
   copy it back to host and again to the GPU */
energy_point* generate_points_by_resolution_parallel(cube *curr_cube, point3d resolution){

    // data structures used
    point3d t;
    int cnt = 0;
    int dx = resolution.x;
    int dy = resolution.y;
    int dz = resolution.z;
    int x_amt = (curr_cube->max.x - curr_cube->min.x) / dx;
    int y_amt = (curr_cube->max.y - curr_cube->min.y) / dy;
    int z_amt = (curr_cube->max.z - curr_cube->min.z) / dz;
    curr_cube->point_amt = x_amt*y_amt*z_amt;
    point2d *dev_limits;
    energy_point *dev_points;
    curr_cube->points = (energy_point *) malloc(curr_cube->point_amt * sizeof(energy_point));  //nvcc vuole il cast

    // blocks needed to cover all possible points
    int nblocks = (curr_cube->point_amt)/MAX_THREADS+1;
    // allocation of data structures for GPU
    hipMalloc( (void**) &dev_limits, curr_cube->N * sizeof(point2d));
    hipMalloc( (void**) &dev_points, curr_cube->point_amt * sizeof(energy_point));
    // copy of limits array
    hipMemcpy(dev_limits, curr_cube->limits, curr_cube->N * sizeof(point2d), hipMemcpyHostToDevice);
    initialize_points<<<nblocks,MAX_THREADS>>>(dev_limits, dev_points, x_amt, y_amt, z_amt, curr_cube->min.x, curr_cube->min.y, curr_cube->min.z, dx, dy, dz, curr_cube->N, curr_cube->min.z);
    //hipMemcpy(curr_cube->points, dev_points, curr_cube->point_amt * sizeof(energy_point), hipMemcpyDeviceToHost);
    // free structures
    hipFree(dev_limits); 

    return dev_points;
}



void free_cube(cube *cu) {
    if(cu->points!=NULL){
        free(cu->points);
    }
    free(cu->limits);
    return;
}

void free_cubes(cube *c_arr, int n) {
    for (int i = 0; i < n; i++) {
        free_cube(&c_arr[i]);
    }
    return;
}

void generate_rays(ray ray_arr[], ray main_ray, int amount) {
    point3d new_start, new_end, new_delta, ang_coeff;
    int new_steps;
    float main_ray_factor;
    float new_length;
    float norm;
    
    ray_arr[0] = main_ray;
    for (int i = 1; i < amount; i++) {
        main_ray_factor = rand_unit();
        new_start.x = main_ray.start.x + main_ray_factor * (main_ray.end.x - main_ray.start.x);
        new_start.y = main_ray.start.y + main_ray_factor * (main_ray.end.y - main_ray.start.y);
        new_start.z = main_ray.start.z + main_ray_factor * (main_ray.end.z - main_ray.start.z);
        ang_coeff.x = rand_unit();
        ang_coeff.y = rand_unit();
        ang_coeff.z = rand_unit();
        norm = sqrt(pow(ang_coeff.x,2) + pow(ang_coeff.y,2) + pow(ang_coeff.z,2));
        new_delta.x = ang_coeff.x / norm;
        new_delta.y = ang_coeff.y / norm;
        new_delta.z = ang_coeff.z / norm;
        new_steps = rand_unit() * main_ray.steps;
        new_end.x = new_start.x + new_delta.x * new_steps;
        new_end.y = new_start.y + new_delta.y * new_steps;
        new_end.z = new_start.z + new_delta.z * new_steps;

        ray_arr[i].start = new_start;
        ray_arr[i].end = new_end;
        ray_arr[i].delta = new_delta;
        ray_arr[i].steps = new_steps;
    }
    return;
}


